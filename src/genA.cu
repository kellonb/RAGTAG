#include "hip/hip_runtime.h"
/*******************************************||********************************************
                               Genetic algorithm optimizer                               *
                                      genA.cu                                            *
Runs iterations of genetic algoirthm to optimize molecular mechanics dihedral parameters * 
              @author James Maier, Kellon Belfon, Chuan Tian                             *
              @lab Carlos Simmerling lab, Stony Brook University                         *
              @version 3.0 2019 Aug                                                      *
********************************************||*******************************************/
/*****************************************************************************************
* 	                ---------------LOAD LIBRARIES-------------                       *  
*****************************************************************************************/
#include <chrono>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/generate.h>
#include <thrust/device_ptr.h>
#include <list>
#include <map>
#include "load.cpp"
#include "parse.cpp"
using namespace std;

/******** Number of threads for a given block, 256 block threads (index 0 to 255) *******/
const int BLOCK_SIZE=256;

#define HANDLE_ERROR(x) x;

/*****************************************************************************************
*                  Defining the six pivotal functions for the genetic algorithm          *
*  (1) mateIt, (2) mutateIt, (3) scoreIt, (4) calcAreas, (5) moveEm, (6) getSumAreas     *
* note: getSumAreas uses two other functions sumEm and sumEmIndex                        *
******************************************************************************************
******************************************************************************************
*                                | function1: mateIt |                                   *
*                                                                                        *
* @purpose creates offspring from a population, generating crossovers according to pCross*
* @param Vs a global array of all the parent and child genomes (Amplitude parameters)    *
* @param ptrs array of pointers from logical indices to actual indices into Vs for       * 
*        each individual                                                                 *
* @param areas the probabilities for choosing each individual for mating                 *
* @param sumArea pointer to the sum of all the individual areas                          *
* @param rands array of random numbers for crossover                                     *         
* @param pCross probability that crossover occurs                                        *               
* @param pSize number of individuals in the population (possible amplitudes solutions)   *
* @param genomeSize number of genes in a genome (number of dihedral * periodicity)       *
*****************************************************************************************/

__global__ void mateIt(float *Vs, int *ptrs, const float *areas, const float *sumArea, 
        const float *rands, const float pCross, const int pSize, const int genomeSize)
{
  /* figure out index for threads  blockId.x is the index for blocks, 
     blockDIM.x is the elements per blocks (# of threads in a block)
     threadIdx is the index for threads */
  int i=blockIdx.x * blockDim.x + threadIdx.x;

  /* random numbers for crossover */
  int randi=i*3;

  /* multiply i by 2, as we will have 2 parents and 2 offspring using a left bitwise 
  (<<) by 1*/
  i<<=1;

  /* if we're in the population (sometimes warps may go past, don't waste threads) */ 
  if (i<pSize) {
    int parent[2];
    int j;
  /* figure out parents */
    parent[0]=parent[1]=-1;
  /* find parent where cumulative (cum) area (A) is less than random target (tgt) area
    selection of parents depends on cumulative probability being less than the 
    random probabilities (random numbers). 
    The random probabilities (tgtA) is random numbers multiply by sum of all the 
    individual probabilities*/
   
    float cumA=0.0f, tgtA=rands[randi++]* *sumArea; //tgtA random number from 0 to the sumArea
    while(cumA<=tgtA){
      ++parent[0];
      cumA+=areas[ptrs[parent[0]]/genomeSize]; // areas (probabilities) is based on mWo option 
      /* rands[randi-1] is the index back to zero since it is the first set of parents */
    }
#if DEBUG>2
    printf("rands[%d] ; %f ; %f=%f * %f\n",randi, cumA, tgtA, rands[randi-1], *sumArea);
    printf("first parent\n");
#endif
    /* This substract 1st parent area from sum of area  */
    cumA=0.0f; tgtA=rands[randi++]* (*sumArea-areas[ptrs[parent[0]]/genomeSize]); 
    while (cumA<=tgtA){
      ++parent[1];
      if (parent[1]==parent[0])  //Ensure you don't pick the same parents
        ++parent[1];
      cumA+=areas[ptrs[parent[1]]/genomeSize];
    }
#if DEBUG>2
    printf("Make offspring %d from %d and %d (%f=%f*(%f-%f)) %d\n", i, parent[0], 
       parent[1], tgtA, rands[randi-1], *sumArea, areas[ptrs[parent[0]]/genomeSize], randi);
#endif
    /* add offset of pSize to i because it is an offspring (next population) */
    i+=pSize;
    /* use ptrs to get indices into Vs */
    int i0=ptrs[i], i1=ptrs[i+1];
    parent[0]=ptrs[parent[0]];
    parent[1]=ptrs[parent[1]];
    /* set j to index for the next set of Vs */
    j=i0+genomeSize;
    /* put parent[0], parent[1], and i1 relative to i0, so we can just add i0 for index */
    parent[0]-=i0;
    parent[1]-=i0;
    i1-=i0;
    /* start with crossover pt at the end (no crossover) */
    int crossPt=j;
    /* check if we need to do crossover, 
       only do crossover if random number is less than pCross */
    if(rands[randi]<pCross){
      crossPt=i0+1+(int)(rands[randi]/pCross*(float)(genomeSize-1));
    }
    while(i0<crossPt){
      /* load next bit from parent and increment i */
      Vs[i0]=Vs[parent[0]+i0];
      Vs[i1+i0]=Vs[parent[1]+i0];
      ++i0;
    }
    while(i0<j){
      Vs[i0]=Vs[parent[1]+i0];
      Vs[i1+i0]=Vs[parent[0]+i0];
      ++i0;
    }  //end of while loop
  } // end of if i<pSize loop 
}

/*****************************************************************************************
                                | function 2: mutateIt |

 * @brief introduces mutations to the genomes in Vs, according to probability pMut, 
    with a max perturbation of max
 *
 * @param Vs a global array of all the parent and child genomes
 * @param ptrs array of pointers from logical indices to actual indices into Vs for
     each individual
   @param rands array of random numbers
 * @param pSize number of individuals in the population
 * @param pMut probability that a mutation occurs, evaluated for each gene
 * @param max maximum perturbation to an allele
 * @param genomeSize number of genes in a genome
*******************************************************************************************/

__global__ void mutateIt(float *Vs, int *ptrs, const float *rands, const int pSize, const float pMut, const float max, const int genomeSize)
{
  /* figure out index */
  int i=blockIdx.x * blockDim.x + threadIdx.x;
  if(i<pSize){
    // get index into random number array
    int r=i*genomeSize;
    // bounds for the begnining of the chromosome of Vs
    i=ptrs[i];
    // bounds for the end of the chromsome of Vs
    int j=i+genomeSize;
    // want random numbers from [-max, max). will subtract max later
    float scale=2.0f*max/pMut;
    while(i<j){
      // if random number is less than the probability of mutation then 
      if(rands[r]<pMut){
         // mutate the amplitude(Vs) by adding perturbation based on max, random number and pMut
        Vs[i]+=rands[r]*scale-max;
       }
       ++i;
       ++r;
    } // end of while loop
  } 
}

/************************************************************************************************
                                | function 3: scoreIt | 

 * @brief calculates a score indicating the closeness of fit for each individual/chromosome
   (set of parameters) against the training set
 * @param scores score for each conformation, calculated here, output array
 * @param areas weighting for each conformation, no longer need
 * @param Vs a global array of all the parent and child genomes (amplitudes)
 * @param ptrs array of pointers from logical indices to actual indices into Vs for each individual
 * @param tset training set
 * @param tgts targets for training
 * @param wts weights of each point in the training set
 * @param breaks breaks in training set, where different data should not be compared across breaks
 * @param nConf number of conformations in training set
 * @param pSize number of individuals in the population
 * @param genomeSize number of genes in a genome
 * @param xx space to store energy differences for each conformation with test parameters
************************************************************************************************/

__global__ void scoreIt(float *scores, float *areas, const float *Vs, const int *ptrs, const int *ptrsV, const int *ptrsT, const int *ptrsD, const int *allFginDs, const int *nVperFg, const float *tset, const float *tgts, const float *wts, const int *breaks, const int nConf, const int pSize, const int trainingSize, const int genomeSize, const int nFg, const int *nCosperFg, float *xx )
{
  // i represent a chromosome , a set of amplitude parameters, this function will be done for each i (chromosome) at the same time
  int i=blockIdx.x * blockDim.x + threadIdx.x;
  if(i<pSize){
    float *x=xx+i*nConf;  // for the error of each conformation
    // get reference to score, S is the AAE 
    float *S=scores+i;
    // set score to 0
    *S=0.0f;
    // accumulate little s (AAE) for each set
    float s;
    int t;
    int i0;
    /* start at break 0 */
    int b=0;
    /* loop over conformations c */
    int c=0;
    int d=0; // index into dataset
    int fg,tg,beg,end;
    int tindx;
    int pt = ptrs[i]; // set the pointer index into the first element of Vs array
    while(c<nConf){
      //s is the sum of REE 
      s=0.0f;
      /* loop only over in conformations within a dataset */
      while(c<breaks[b+1]){
        /* start with delta E (tgts) for a given conformation (c) within a break; see load.cpp 
          conf (c) goes through until it reach a break. the loop will set delta E */

        // get first index in genome
        i0=pt;
#if DEBUG>2  
        printf("i0: %d ", i0);
#endif     
        // get dE for that conformation 
        x[c]=tgts[c];
        // Get the number of dihedral in the dataset
        // loop throught the dihedrals of a given conformation 
#if DEBUG>2  
        printf("ptrsD ??: ptrsD[d] = %d, ptrsD[d+1] = %d, d = %d\n", ptrsD[d],ptrsD[d+1],d);
#endif     
        tindx=0; //index into the ptrsT array 0 to number of dihedral columns in a given dataset
        for (int dih=ptrsD[d];dih<ptrsD[d+1];dih++,tindx++){
          //Get the fitting group for that dihedral 
          fg=allFginDs[dih];
#if DEBUG>2  
          printf("Fitting group = %d for dih index %d\n", allFginDs[dih], dih);
#endif     
          //get the index into Vs and tset
          beg=i0+ptrsV[fg];
          end=beg+nVperFg[fg];
          tg=ptrsT[(c*trainingSize)+tindx]; //index into prtsT
          t=(c*trainingSize)+tg;
#if DEBUG>2  
          printf("beg = %d, end = %d, tg = %d, tindx = %d t = %d \n", beg,end,tg,tindx,t);
#endif     
          //loop through the number of cosines 
          for (int i=beg;i<end;i++,t++) {
             /* subtract contributions from each parameter for conformation c for each conformation 
             e.g deltaE - cos (dihedral * periodicity) * parameter generated from chromosomes 
	     Therefore, it is delta E - sum of cosines for each dihedral */
            x[c]-=Vs[i] * tset[t]; // Vs* tset is cos(n * dih)
#if DEBUG>2  
            printf("scoreIt: i = %d, c = %d, dih = %d, beg = %d, end = %d, t = %d, x[c] = %f,  Vs[i] = %f, tset[t] = %f \n",i,c,dih,beg,end,t,x[c],Vs[i],tset[t]);
#endif     
          }

        }
        /* add differences in this error from all other errors */
#if DEBUG>2
        printf("outside loopscore for x[c] = %f\n", x[c]);
#endif
        for(int c2=breaks[b];c2<c;c2++){
#if DEBUG>2
          printf("In loop score for x[c] = %f\n", x[c]);
          printf("%d - %d\n",c,c2); //print the pairs index
#endif
          // calculate the absolute error for each pairs 
          float err=x[c]-x[c2];
          // sum the absolute of the errors (err) - -err = + err ; +err = +err
          //s+=(err<0.0f?-err:err); //ternary operator, condition is err < 0.0; if true err is negative, if false error is positive 
          s+=abs(err); 
        }
        /* next conformation */
        ++c;
      } 
      /* add little error to big error S, weighted by number of pairs, wt  is 2 / nconf*(nconf-1) */
      *S+=s*wts[b];
      /* go to next breakpoint (data set) */
      ++b;
      ++d;
    }
  } //end if in Psize
}

/**************************************************************************************************
*                                 | function 4: calcAreas |                                       *
*                                                                                                 *
*     calculates the areas (the probability) each individual has of mating                        *
*___________________________________Parameters____________________________________________________*
* @param scores scores for each individual (set of parameters)                                    *
* @param areas fitness for each individual, in terms of probability of mating                     *
* @param ptrs array of pointers from logical indices to actual indices into Vs for each individual*
* @param pSize number of individuals in the population                                            *
* @param genomeSize number of genes in a genome                                                   *
**************************************************************************************************/

__global__ void calcAreas(float *scores, float *areas, const int *ptrs, const int pSize, const int genomeSize, const int weight_flag, float temperature) {
  int i=blockIdx.x * blockDim.x + threadIdx.x;
  float b_k = 0.001987204; // kcal/mol/K boltzmann constant
  float kt = b_k * temperature; // K 
  if(i<pSize){
    // if the weight flag is 1 then use a heavy weight 
    if (weight_flag==1){
      areas[ptrs[i]/genomeSize]=__expf(-scores[i]/scores[0]);
    }
    // use 1/1+si
    else if (weight_flag==2){
      areas[ptrs[i]/genomeSize]= 1/(1 + scores[i]);
    } 
    // use same as 1 but with kt instead so you can adjust the probabilities
    else if (weight_flag==3){
      areas[ptrs[i]/genomeSize]=__expf(-scores[i]/kt);
    }  
  }
}
/*****************************************************************************************
*                                | function 5: moveEm |
*
* @brief simple helper function for copying data from oldF, oldI to neWF, newI
*
* @param newF pointer to new float array
* @param newI pointer to new int array
* @param oldF pointer to old float array
* @param oldI pointer to old int array
* @param N number of floats/ints to copy
*****************************************************************************************/
__global__ void moveEm(float * newF, int *newI, float *oldF, int *oldI, int N) {
  int i=blockIdx.x * blockDim.x + threadIdx.x;
  if(i<N){
    newF[i]=oldF[i];
    newI[i]=oldI[i];
  }
}
/******************************| function 5 ends |***************************************/

/*****************************************************************************************
                   | sumEm and sumEmIndex : helper function for getSumAreas |

* @brief performs a sum of each successive pair of N numbers in source and stores the sums 
         in sums. intended to be run multiple times to sum over a whole array. if N is odd, 
         the last sum index will be N/2-1 and contain the sum of the last 3 numbers
*
* @param sums where to store the sums
* @param source where to get the numbers to sum together
* @param N the dimension of source
*
* @return                        ********************************************************/

__global__ void sumEm(float *sums, float *source, int N){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=(i<<1);
  if(j+3<N)sums[i]=source[j]+source[j+1];
  else if(j+3==N) sums[i]=source[j]+source[j+1]+source[j+2];
  else if(j+2==N) sums[i]=source[j]+source[j+1];
}

/*
* @brief performs a sum of pairs of N numbers in source, using locations indicated 
         by pointers. pointers has indices multiplied by genomeSize. intended to be
         run multiple times to sum over a whole array. if N is odd, the last sum index 
         will be N/2-1 and contain the sum of the last 3 numbers
*
* @param sums where to store the sums
* @param source an array where to get the numbers to sum together
* @param N the dimension of source
* @param ptrs the indices to use when gathering pairs for summation
* @param genomeSize the number by which the indices in ptrs are scaled
*
* @return 
*/
__global__ void sumEmIndex(float *sums, float *source, int N, const int *ptrs, const int genomeSize){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=(i<<1); // j = i*2 (mutiplication using a left bitwise shift)
  if(j+3<N)sums[i]=source[ptrs[j]/genomeSize]+source[ptrs[j+1]/genomeSize];
  else if(j+3==N) sums[i]=source[ptrs[j]/genomeSize]+source[ptrs[j+1]/genomeSize]+source[ptrs[j+2]/genomeSize];
  else if(j+2==N) sums[i]=source[ptrs[j]/genomeSize]+source[ptrs[j+1]/genomeSize];
#if DEBUG>1
  if(j+2<=N)printf(" %d:%f",i,sums[i]);
#endif
}
/*******************************| end of helper function |*******************************/
/*****************************************************************************************
*                                | function 6: getSumAreas |                             * 
*                        ---------uses sumEmIndex and sumEM--------                      *
*                                                                                        *
* @brief get sum of all areas                                                            *
* @param areas_d pointer to areas on device                                              *
* @param ptrs_d pointer to indices for each individual in population                     *
* @param pSize population size                                                           *
* @param temp_d pointer to temporary array on device                                     *
* @param genomeSize number of alleles in genome                                          *
*****************************************************************************************/

float *getSumAreas(float *areas_d, int *ptrs_d, int pSize, float *temp_d, const int & genomeSize){
  int dim=pSize; //Set dim to pSize
  int offset=0;

  // return an array of sums (temp_d), sum up the probabilities in areas_d array
  sumEmIndex <<<((dim>>1)+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (temp_d, areas_d, dim, ptrs_d, genomeSize);
  pSize >>= 1;  
  while((dim>>=1)>1){  // while pSize/2 is greater than 1: Keep dividing (1/2 psize) by 2  
    offset^=pSize;  //bitwise XOR offest is 1/2 pSize then 0, then 1/2 pSize, then 0...
    // doing this switch the source to be (temp+pSize/2) then the source changes to (temp_d+0), then back and forth
    sumEm <<<((dim>>1)+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (temp_d+offset, temp_d+(offset^pSize), dim);
  }
  return temp_d+offset;
}


/*
///////////////////////////////////////////////////////                 `
//////////////////////////////////                                       `
/////////////////////                                                  |   | 
/////////////                                                     ~ ~ ~ ~ ~ ~ ~
////////                                                         |              |
/////                                                        ____|              |____  
///                                                         |                        | 
//                                                       ___|          J.M           |___
/                                                       |              K.B               |
/                              PROGRAM BEGINS HERE      |              C.T               |
*****************************************************************************************/

/*****************************************************************************************
argc is a vairable with the number of arguments passed to GenA
argv is a vector of strings representing the the arguments the GenA takes
input file: parametersfitting data using the following format:
 _____________________________________________________________________        
|-<dihedral> <AMBER atom type for dihedral 1> -Fg_0 periodicities     |
|-<dihedral> <AMBER atom type for dihedral 2> -Fg_1 periodicities     |
|<name of data set> <weights <dihedral 1> <dihedral 2>  ndih>         |
| <dihedral 1 value> <dihedral 2 value> <E_QM> <E_MM>                 |
| <dihedral 1 value> <dihedral 2 value> <E_QM> <E_MM>                 |
|                    ...                                              |
|/                                                                    | 
|<name of data set> <weights <dihedral 1> <dihedral 2> ndih>          |
| <dihedral 1 value> <dihedral 2 value> <E_QM> <E_MM>                 |
| <dihedral 1 value> <dihedral 2 value> <E_QM> <E_MM>                 |  
|                   ...                                               |
|/                                                                    |  
|_____________________________________________________________________|

<dihedral> is the name of dihedral e.g phi, psi, chi1, chi2, chi3, etc
ndih> is the number of dihedral in the dataset
<AMBER atom type for dihedral 1> e.g chi1 is N -CX-2C-2C for Met, get from frcmod file
<name of data set> is any name, e.g Metalpha, Metbeta, Metcharge
<dihedral 1 value> this is the dihedral value (deg) of the optimized QM structures 
     e.g 105.62
<E_QM> the QM energy of conformation i with restraint dihedral
<E_MM> the MM energy of conformation i with with zeroed dihedral parameters in the 
       frcmod
... repeat for all conformations within a break 
/ (refer to as break (brk))
a break seperate conformations that are different database
    e.g alpha backbone, beta backbone, charge amino acids
                                  GOODLUCK!!!
                                  [ O    O ]
                                  [    b ' ]
                                  [  ----- ]
contact: kellonbelfon@gmail.com with RAGTAG title for help
*****************************************************************************************/

int main(int argc, char *argv[]){

  /* start the timer */
  auto t1=std::chrono::high_resolution_clock::now();
  /*specify the string name of the savefile, scorefile, loadfile etc */
  std::string saveFile, loadFile, scoreFile, logFile, frcmodFile, inputFile, fitFile;
  /* genetic algorithm parameters initiated */
  int pSize, nGen, rseed, peng, ncp, nCos, nChrom, nDih, nFg, mWo;
  float pMut, max, pCross, keep, tempe;
  /* getting the filenames from the commands -r, -c, -s, -o, -f -y -a */
  for (int i=1;i<argc;i++){
    if(i+1<argc){
      if(argv[i][0]=='-'&&argv[i][1]=='r')saveFile=argv[++i];  //file that save amplitudes parameter (Vs)
      else if(argv[i][0]=='-'&&argv[i][1]=='c')loadFile=argv[++i]; //file with Vs for restart or from other forcefields
      else if(argv[i][0]=='-'&&argv[i][1]=='s')scoreFile=argv[++i]; // file that save the scores
      else if(argv[i][0]=='-'&&argv[i][1]=='f')frcmodFile=argv[++i]; //file that save frcmod file
      else if(argv[i][0]=='-'&&argv[i][1]=='o')logFile=argv[++i]; //file that save outputs 
      else if(argv[i][0]=='-'&&argv[i][1]=='i')inputFile=argv[++i]; // input file with dihedral info
      else if(argv[i][0]=='-'&&argv[i][1]=='y')fitFile=argv[++i]; // file with and idea of how your target energy change
    }
  }
  /* open the output file which is the log file */
  std::ofstream logfile;
  logfile.open (logFile.c_str(), ios::out);
  /* open the score file to store scores */
  std::ofstream scorefile;
  scorefile.open (scoreFile.c_str(), ios::out); 
  scorefile << "#Generation" << std::setw(14) << "Chromosomes" << std::setw(12) << "Scores" << std::setw(14) << "areas\n";
 
  /* Now load genA parameters, from the parmfile -p  */
  for (int i=1;i<argc;i++){
    if(i+1<argc){
      if(argv[i][0]=='-'&&argv[i][1]=='p'){
      ConfigFile cfg(argv[++i]); //file that has the genetic algorithm parameters
      // check if keys exixt and set a message to the user that we are using the default 
      if (!(cfg.keyExists("pSize"))) std::cout << "pSize was not specified, using default of 2000\n";  
      if (!(cfg.keyExists("nGen"))) std::cout << "nGen was not specified, using default of 1000\n";  
      if (!(cfg.keyExists("pMut"))) std::cout << "pMut was not specified, using default of 0.01\n";  
      if (!(cfg.keyExists("max"))) std::cout << "max was not specified, using default of 0.5\n";  
      if (!(cfg.keyExists("pCross"))) std::cout << "pCross was not specified, using default of 0.8\n";  
      if (!(cfg.keyExists("peng"))) std::cout << "peng was not specified, using default of 10\n";  
      if (!(cfg.keyExists("ncp"))) std::cout << "ncp was not specified, using default of 2\n";  
      if (!(cfg.keyExists("keep"))) std::cout << "keep was not specified, using default of 0.2\n";  
      if (!(cfg.keyExists("nDih"))) std::cout << "nDih was not specified, using default of 1\n";  
      if (!(cfg.keyExists("nFg"))) std::cout << "nFg was not specified, using default of 1\n";  
      if (!(cfg.keyExists("mWo"))) std::cout << "mWo was not specified, using default of 1\n";  
      // Retreive the value of keys 
      pSize = cfg.getValueOfKey<int>("pSize", 2000);
      logfile << "Population Size (pSize): " << pSize << "\n\n";
      nGen = cfg.getValueOfKey<int>("nGen", 1000);
      logfile << "Number of Generations (nGen): " << nGen << "\n\n";
      pMut = cfg.getValueOfKey<float>("pMut", 0.01);
      logfile << "Probability of Mutations (pMut): " << pMut << "\n\n";
      max = cfg.getValueOfKey<float>("max", 0.5);
      logfile << "Maximal permissible mutation (max): " << max << "\n\n";
      pCross = cfg.getValueOfKey<float>("pCross", 0.8);
      logfile << "Probability of crossover (pCross): " << pCross << "\n\n";
      rseed = cfg.getValueOfKey<int>("rseed", 314245);
      logfile << "Random seed (rseed): " << rseed << "\n\n";
      peng  = cfg.getValueOfKey<int>("peng", 10);
      logfile << "Print scores every  " << peng << "generations (peng)\n\n";
      ncp  = cfg.getValueOfKey<int>("ncp", 2);
      logfile << "Print scores of only " << ncp << " chromosomes every peng \n\n";
      nCos = cfg.getValueOfKey<int>("nCos", 4);
      logfile << "Periodicity (nCos): " << nCos << "\n\n";
      keep = cfg.getValueOfKey<float>("keep", 0.2);
      logfile << "We will use " << keep << " for the elitist regime\n\n"; 
      nDih = cfg.getValueOfKey<int>("nDih", 1);
      logfile << "Number of dihedral(s) (nDih): " << nDih << "\n\n";
      nFg = cfg.getValueOfKey<int>("nFg", 1);
      logfile << "Number of Fitting groups (nFg): " << nFg << "\n\n";
      mWo = cfg.getValueOfKey<int>("mWo", 1);
      logfile << "Mating weight option flag " << mWo << "\n\n";
      // it the mating weight option is 3 then read temperature 
      if (mWo==3) {
         tempe = cfg.getValueOfKey<int>("tempe", 298.0);
         logfile << "Temperature (K)  " << tempe << "\n\n";
      }
       if(!loadFile.empty()) {
        nChrom = cfg.getValueOfKey<int>("nChrom", 100);
        logfile << "Number of chromosome reported is : " << nChrom << "\n\n";
        }
      }
    } 
  }
 
  /* initializing GPU (_d) and CPU arrays */ 
  hipError_t error;
  size_t nRands;
  hiprandGenerator_t gen;
  float *Vs, *Vs_d, *rands, *rands_d, *tset, *tset_d, *tgts, *tgts_d, *wts, *wts_d, *xx_d;
  float *scores, *scores_d, *areas, *areas_d, *EMM0;
  int genomeSize, trainingSize, g, totdih, *ptrs_d, *ptrs, N, nConf=0, nDataset=0, *breaks, *breaks_d, nBreaks; 
  int *ptrsT, *ptrsV, *ptrsD, *ptrsT_d, *ptrsV_d, *ptrsD_d, *allFginDs, *allFginDs_d;
  int *nCosperFg, *nCosperFg_d, *nVperFg, *nVperFg_d, *nDihperDs, *nDihperDs_d, *DihFgindx;
  int save=pSize*keep; //save is number of chromosome we will keep as elitist

/***************************| load data from load.cpp |***********************************
*  check load.cpp for this section                                                       *
*  map is a way to create a dictionary, correction map is an array with key              * 
*****************************************************************************************/

/* initiating container with key and values name correctionMap */
  std::map<std::string,DihCorrection> correctionMap;

/* input file open, with dihedral info */ 
  std::ifstream inputfile;
  inputfile.open (inputFile.c_str(), std::ios::in);
 
/* load in arrays generated from load.cpp, check it out for further comments */
  load(inputfile, &tset, &ptrsV, &ptrsT, &ptrsD, &allFginDs, &nDihperDs, &tgts, &wts, &nConf, &nDataset, &breaks, &nBreaks, &trainingSize, &genomeSize, 
       correctionMap, &nVperFg, &nCosperFg, nCos, nFg, nDih, &totdih, &DihFgindx, &EMM0);
  logfile << "Input file loaded ('_')" << "\n\n";
/****************************************************************************************/

/*************************| memory allocation |*******************************************
*   Declare and allocate host and device memory, copy data arrays from CPU host 
       (breaks,tset,                                                                 
*     tgts,wts) to device GPU (breaks_d, etc)                                        
*****************************************************************************************/

#if DEBUG && 0
  for(int i=0;i<nConf;i++){
    for(int j=0;j<trainingSize;j++)
      std::cerr << ' ' << tset[i*trainingSize+j];
    std::cerr << std::endl;
  }
  std::cerr << tgts[0] << ' ' << tgts[1] << ' ' << tgts[2] << ' ' << tgts[3] << std::endl;
  std::cerr << "first hipMalloc, " << nBreaks << " breaks" << std::endl;
#endif

  // memory allocation onf the GPU
  hipMalloc(&nCosperFg_d, nFg*sizeof(int));
  hipMalloc(&ptrsV_d, nFg*sizeof(int));
  hipMalloc(&ptrsT_d, nConf*trainingSize*sizeof(int));
  hipMalloc(&ptrsD_d, (nDataset+1)*sizeof(int));
  hipMalloc(&nDihperDs_d, (nDataset+1)*sizeof(int));
  hipMalloc(&allFginDs_d, totdih*sizeof(int));
  hipMalloc(&nVperFg_d, nFg*sizeof(int));
  // Some cuda copies, here TODO: Copy all at the same time, to reduce time
  hipMemcpy(ptrsV_d, ptrsV, nFg*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(ptrsT_d, ptrsT, nConf*trainingSize*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(ptrsD_d, ptrsD, (nDataset+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(nDihperDs_d, nDihperDs, (nDataset+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(allFginDs_d, allFginDs, totdih*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(nVperFg_d, nVperFg, nFg*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(nCosperFg_d, nCosperFg, nFg*sizeof(int), hipMemcpyHostToDevice);
 
  hipMalloc((void **)&breaks_d, nBreaks*sizeof(int));
  hipMalloc((void **)&tgts_d, (nBreaks-1+nConf*(1+trainingSize))*sizeof(float));
  wts_d=tgts_d+nConf;
  tset_d=wts_d+nBreaks-1;

#if DEBUG
  std::cerr << "COPY" << std::endl;
#endif

/* Copying over the arrays from the CPU to GPU
nbreaks is the # of dataset + 1. e.g if you are doing alpha and beta backbone set then nbreaks=3
genomesize is the # of fitting dihedral * periodicity, e.g 3 set of dihedral * 4 periodicity = 12
nconf is the # of conformations you are fitting
tgts is (E_QMi-E_MMi) + (E_MMref-E_QMref) for each conformation, which = nconf, see load.cpp
tset is the cos(dih*periodicity) for 4 periodicity for a dihedral for each conformation
so 20 conf will give tgts of 20 (nconf) * 12 (# of dih * periodicity) = 120 
*/
  hipMemcpy(breaks_d, breaks, nBreaks*sizeof(breaks[0]), hipMemcpyHostToDevice);
  if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error:(Memcpy breaks) %s\n", hipGetErrorString(error));}
  hipMemcpy(tset_d, tset, nConf*trainingSize*sizeof(float), hipMemcpyHostToDevice);
  printf("trainingSize is %d after cuda copy\n", trainingSize);
  if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error:(Memcpy tset) %s\n", hipGetErrorString(error));}
  hipMemcpy(tgts_d, tgts, nConf*sizeof(float), hipMemcpyHostToDevice);
  if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: (Memcpy tgts) %s\n", hipGetErrorString(error));}
  hipMemcpy(wts_d, wts, (nBreaks-1)*sizeof(*wts), hipMemcpyHostToDevice);
  if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: (Memcpy wts) %s\n", hipGetErrorString(error));}

/**********************| initiate GPU blocks and # of random variable |*************************** 
*          we need randoms, new pop 3xcrossover, genomeSizexmut                                  *    
*        genome size is the number of genes which is all the parameters,                         *
*   e.g for 4 periodicity and three dihedral fitting, then genomesize will be 4 * 3 = 12         *
*   nRands is number of randoms we need for each set of parameters                               *
*   e.g if psize (population size) is 10, then number of random number we will need is           *
*                   (3+(# of periodicity x # of dihedral)) * psize                               *
* so for 4 periodicity and 3 dihedral fitting (chi1 chi2 chi3), then nRands = 3+12 * 10 = 150    *
*________________________________________________________________________________________________*  
*  nBlocks is dependent on the population size, it is use to figure out how many GPU blocks      *
*  we need to initialize the arrays for calculations. Each block has 256 threads.                *
*  one thread represent one individual (chromosome with soln parameters) from the population     *
*   e.g population size of 2000 will require (2000+256-1)/256 = 8.81 => 8 blocks                 *
*                                                                                                *
*************************************************************************************************/
  nRands=(3+genomeSize)*pSize;
  int nBlocks=(pSize+BLOCK_SIZE-1)/BLOCK_SIZE;

/*******************************| initializing host and device variables|************************
*         N (bitwise operation below) is the pSize (1st input) multiply by 2;                   *
*       initiating the chromosomes  which have the solns                                        *
************************************************************************************************/

  rands=(float *)malloc(nRands*sizeof(float));
  N=(pSize<<1);
  HANDLE_ERROR(hipMalloc((void **)&Vs_d, (N*(genomeSize+4)+pSize*nConf+nRands)*sizeof(float)));
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: (Malloc Vs_d) %s\n", hipGetErrorString(error));}
  rands_d=Vs_d+N*genomeSize;
  scores_d=rands_d+nRands;
  areas_d=scores_d+(N<<1);
  xx_d=areas_d+(N<<1);
  scores=(float *)malloc(sizeof(*scores)*N);
  float *scores_ds[2];
  scores_ds[0]=scores_d;
  scores_ds[1]=scores_d+N;
  printf("GENOMESIZE: %d \n", genomeSize);

  // allocate memory to host Vs (amplitudes or barrier height for the cosine function)
  Vs=(float *)malloc(N*genomeSize*sizeof(float));
  areas=(float *)malloc(N*sizeof(float));
  /* allocate the memory space to hold array of pointers (prts) of size N (2*pSize)
  these pointers point to the individuals (chromosome) in the population */
  ptrs=(int *)malloc(sizeof(int)*N);
  ptrs[0]=0;
  for(g=1;g<N;g++)ptrs[g]=ptrs[g-1]+genomeSize;
  HANDLE_ERROR(hipMalloc((void **)&ptrs_d, N*2*sizeof(int)));
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: (Malloc ptrs_d) %s\n", hipGetErrorString(error));}
  int *ptrs_ds[2];
  ptrs_ds[0]=ptrs_d;
  ptrs_ds[1]=ptrs_d+N;
  hipMemcpy(ptrs_d, ptrs, sizeof(int)*N, hipMemcpyHostToDevice);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: (Memcpy ptrs_d) %s\n", hipGetErrorString(error));}
  int curList=0;

/* thrust is a c++ template library for CUDA similar to STL it have two containers: 
       thrust::host_vector<type> and thrust::device_vector<type>
  The containers make common operations such as hipMalloc, hipFree, hipMemcpy, more concise
  e.g thrust::host_vector<int> vec_h(2) will allocate host vector with 2 elements
    thrust::device_vectore<int> vec_d = vec_h will copy host vector to device
  This will allow you to directly manipulate device values from the host
    so vec_d[0] = 5; can be done from host  and once you output vector memory is 
    automatically released 
   it have a few algorithms, we use thrust::sort(), */
  thrust::device_ptr<int> dPtrs(ptrs_d), dPtrs_save(ptrs_d+save);
  thrust::device_ptr<float> dScores(scores_d), dVs(Vs_d);
  thrust::device_ptr<float> dScores_save(scores_d+save),
                            dScores_pSize(scores_d+pSize),
                            dScores_N(scores_d+N);


/**************************| Create a random generator |********************************************
*hiprandSetPseudoRandomGeneratorSeed takes two parameters (1) the generator (gen) & (2) seed value  *
* seed value # is used to initialize the generator and control the set of random numbers;          *
* same seed will the give same set of random numbers of the psuedorandom generator                 *
* rseed is the random number specified from the 6th input)                                         *
*__________________________________________________________________________________________________*
*    hiprandGenerateNormal take 5 parameters:                                                       * 
*  (1) generator - Generator to use                                                                *
*  (2) outputPtr - Pointer to device memory to store CUDA-generated results,                       *
                or Pointer to host memory to store CPU-generated resluts                           *
*  (3) num - Number of floats to generate                                                          *
*  (4) mean - Mean of normal distribution                                                          *
*  (5) stddev - Standard deviation of normal distribution                                          *
* Results are 32-bit floating point values with mean and standard deviation.                       * 
***************************************************************************************************/

  // create the generator name gen
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  // initiate the generator with the random seed (rseed) for natural distribution of random numbers 
  hiprandSetPseudoRandomGeneratorSeed(gen, rseed);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (seed)\n", hipGetErrorString(error));}
  // Vs_d is the amplitudes which is random numbers but can be overwritten with preloaded Vs
  hiprandGenerateNormal(gen, Vs_d, N*genomeSize, 0, 1);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (normal)\n", hipGetErrorString(error));}

#if DEBUG
  hipMemcpy(Vs, Vs_d, sizeof(float)*genomeSize*N, hipMemcpyDeviceToHost);
  /// print the three Vs from the first two chromosomes. 
  std::cout << "random Vs, created on GPU" << std::endl;
  for(int i=0;i<1;i++){
    std::cout <<  Vs[ptrs[i]] << " " << Vs[ptrs[i]+1] << " " << Vs[ptrs[i]+2] << std::endl;  
  }    
#endif

  /*****  if we have a load file copy Vs (amplitude parameters) from the loaded file and populate Vs ***********/
  if(!loadFile.empty()) {
    std::ifstream loadfile;
    loadfile.open (loadFile.c_str(), std::ios::in);
    // copy the random Vs to add previous chromosome of nChrom
    hipMemcpy(Vs, Vs_d, sizeof(float)*genomeSize*N, hipMemcpyDeviceToHost);
    if (loadfile.is_open()) {
      for (int i=0;i<nChrom;i++) {
        for (int j=0;j<genomeSize;j++) {
          loadfile >> Vs[ptrs[i]+j]; 
        }
      }
    }
    // print the two Vs from the first two chromosomes, to ensure your Vs were loaded. 
    logfile << "Here is your loaded Vs(amplitudes) for first three chromosomes: \n\n" << std::endl;
    for(int i=0;i<3;i++){
      for(int j=0;j<genomeSize;j++){
        logfile <<  Vs[ptrs[i]+j] << " ";  
      }
      logfile << "\n";
    }      

    // copy loaded Vs  to the GPU and overwrite random Vs. If user only create two chromosomes or 
    // previous Vs then the rest of the chromosome will be random 
    hipMemcpy(Vs_d, Vs, N*genomeSize*sizeof(*Vs), hipMemcpyHostToDevice);// copy to GPU 
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: (loadingVs) %s\n", hipGetErrorString(error));}
  }

#if DEBUG
  // check to see if Vs was transfer to gpu successful 
  /// print the three Vs from the first two chromosomes. 
  std::cout << "loaded Vs" << std::endl;
  for(int i=0;i<1;i++){
    std::cout <<  Vs[ptrs[i]] << " " << Vs[ptrs[i]+1] << " " << Vs[ptrs[i]+2] << std::endl;  
  }    
  hipMemcpy(Vs_d, Vs, pSize*genomeSize*sizeof(*Vs), hipMemcpyHostToDevice);// copy to GPU 
  hipMemcpy(Vs, Vs_d, sizeof(float)*genomeSize*N, hipMemcpyDeviceToHost); // copy back to CPU
  /// print the three Vs from the first two chromosomes. 
  std::cout << "After transfer of loaded Vs to GPU" << std::endl;
  for(int i=0;i<1;i++){
    std::cout <<  Vs[ptrs[i]] << " " << Vs[ptrs[i]+1] << " " << Vs[ptrs[i]+2] << std::endl;  
  }    
#endif
 
/***************************| score of the first set of chromosomes |*******************************
* Here we score the two arrays of parents with solution parameters in the initial population       * 
***************************************** *******************************************************/
    // lauch first kernel to score the initial set of chromsomes (Vs_d) and output scores in scores_ds
    scoreIt <<<(N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (scores_ds[curList], areas_d, Vs_d, ptrs_ds[curList], 
                 ptrsV_d, ptrsT_d, ptrsD_d, allFginDs_d, nVperFg_d, tset_d, tgts_d, wts_d, breaks_d, 
                 nConf, pSize, trainingSize, genomeSize, nFg, nCosperFg_d, xx_d);
    // score of chromosomes outside of psize since we initiated 2 times psize
    scoreIt <<<(N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (scores_ds[curList]+pSize, areas_d, Vs_d, ptrs_ds[curList]+pSize, 
                 ptrsV_d, ptrsT_d, ptrsD_d, allFginDs_d, nVperFg_d, tset_d, tgts_d, wts_d, breaks_d, 
                 nConf, pSize, trainingSize, genomeSize, nFg, nCosperFg_d, xx_d);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (1stscore)\n", hipGetErrorString(error));}
    // print the initial scores based on ncp as Initial, doing this before sorting so as to score the loaded Vs parameters 
    hipMemcpy(scores, scores_ds[curList], sizeof(*scores)*ncp, hipMemcpyDeviceToHost);
    for(int m=0;m<ncp;m++){
      scorefile << std::setw(6) << "Initial" << std::setw(14) << m << std::setw(18) << scores[m]/nDataset << "\n";
    }
    /* sort the scores from each chromosome of the initial population */
    thrust::sort_by_key(thrust::device_pointer_cast(scores_ds[curList]), thrust::device_pointer_cast(scores_ds[curList]+N), thrust::device_pointer_cast(ptrs_ds[curList]));
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (1stsort)\n", hipGetErrorString(error));}
    // print the initial scores based on ncp as -1, doing this after  sorting so we can see how good the best ones are  
    hipMemcpy(scores, scores_ds[curList], sizeof(*scores)*ncp, hipMemcpyDeviceToHost);
    for(int m=0;m<ncp;m++){
      scorefile << std::setw(6) << "Init_after_sort" << std::setw(14) << m << std::setw(18) << scores[m]/nDataset << "\n";
    }

#if DEBUG>2
    hipMemcpy(scores, scores_ds[curList], sizeof(*scores)*N, hipMemcpyDeviceToHost);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n (memcpy scores)", hipGetErrorString(error));}
    hipMemcpy(Vs, Vs_d, sizeof(*Vs)*N*genomeSize, hipMemcpyDeviceToHost);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
    hipMemcpy(ptrs, ptrs_ds[curList], sizeof(*ptrs)*N, hipMemcpyDeviceToHost);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
       /* i is each chromosome, scores[i] is scores, Vs[ptrs[i]] is the amplitude parameters;
         Vs[ptrs[i]]+n specifies the next n amplitude. e.g chromosome i have genomesize amplitude parms 
         e.g  Vs[ptrs[i]]+1 is the amplitude term when the periodicity is 3 for the 1st dihedral being
        fitted, and  Vs[ptrs[i]]+4, the amplitude term when the periodicity is 4 for the 2nd dihedral */
    for(int i=0;i<N;i++){
      std::cerr << i << ": [" << ptrs[i] << "] = " << scores[i] << " {"<<Vs[ptrs[i]]<<" "<<Vs[ptrs[i]+1]<<" "<<Vs[ptrs[i]+2]<<" "<<Vs[ptrs[i]+3]<<"}\n";
    }
#endif

/****************************| Let us begin the iterations through generations |********************

 Genetic algorithm iterations through the number of generations (nGen: 2nd input) 

****************************************************************************************************/

  /* for loop for the generation */
  for(g=0;g<nGen;g++){
    // create an array of random numbers (rands_d) used for mutations and crossover where the number of random #s is nRands 
    hiprandGenerateUniform(gen, rands_d, nRands);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: (GenerateUniform)%s\n", hipGetErrorString(error));}

    // Step2: calculate the probabilities (areas) each individual (chromosome) has of mating 
    calcAreas <<<nBlocks, BLOCK_SIZE>>> (scores_ds[curList], areas_d, ptrs_d, pSize, genomeSize, mWo, tempe);

    // Step3:  mate the individuals (chromosomes,Parent[0],[1]) selected for the next generation 
    mateIt <<<nBlocks, BLOCK_SIZE>>> (Vs_d, ptrs_ds[curList], areas_d, 
          getSumAreas(areas_d, ptrs_ds[curList], pSize, areas_d+N, genomeSize),
          rands_d, pCross, pSize, genomeSize);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (mate)\n", hipGetErrorString(error));}

    //  Step4: mutate individuals generated after mating 
    mutateIt <<<nBlocks, BLOCK_SIZE>>> (Vs_d, ptrs_ds[curList]+pSize, rands_d+pSize*3, pSize, pMut, max, genomeSize);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (mutate)\n", hipGetErrorString(error));}

    // Step5: Score the individuals to select for the next generation 
    scoreIt <<<nBlocks, BLOCK_SIZE>>> (scores_ds[curList]+pSize, areas_d, Vs_d, ptrs_ds[curList]+pSize, 
            ptrsV_d, ptrsT_d, ptrsD_d, allFginDs_d, nVperFg_d, tset_d, tgts_d, wts_d, breaks_d, 
            nConf, pSize, trainingSize, genomeSize, nFg, nCosperFg_d, xx_d);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (score)\n", hipGetErrorString(error));}

    // Step6: Sort the scored chromosomes (individuals) & select for mating for next generation 
      // curList^1 change curList to 1
      // move the scores and pointers to the chromosome for the elitist parents)
    moveEm <<<(save+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (scores_ds[curList^1], ptrs_ds[curList^1], scores_ds[curList], ptrs_ds[curList], save);
     // curList^1 change curList to 0
    // move the scores and pointers to the chromosome for the offsprings
    moveEm <<<(pSize+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (scores_ds[curList^1]+save, ptrs_ds[curList^1]+save, scores_ds[curList]+pSize, ptrs_ds[curList]+pSize, pSize);//nOffspring);
    // curList^1 change curList to 1
     // move the scores and pointers to the chromosome of the left over parent
    moveEm <<<(pSize-save+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (scores_ds[curList^1]+save+pSize, ptrs_ds[curList^1]+save+pSize, scores_ds[curList]+save, ptrs_ds[curList]+save, pSize-save);
    // curList back to 0
    curList^=1;

    /* first sort only the offspring */
#if DEBUG>1
    std::cerr << "Selection sort (" << N << " items, less " << save << ")" << std::endl;
#endif
    thrust::sort_by_key(thrust::device_pointer_cast(scores_ds[curList]+save), thrust::device_pointer_cast(scores_ds[curList]+pSize+save), thrust::device_pointer_cast(ptrs_ds[curList]+save));

    /* second sort  is to sort the elitist parent and the offsprings (psize-save) that fall into pSize */
#if DEBUG>1
    std::cerr << "Rank sort" << std::endl;
#endif
    thrust::sort_by_key(thrust::device_pointer_cast(scores_ds[curList]), thrust::device_pointer_cast(scores_ds[curList]+pSize), thrust::device_pointer_cast(ptrs_ds[curList]));

/****************************************************************************************************
* Here you can print the score of chromosomes (total is 2 x population size) to score file (-s)     *
****************************************************************************************************/
    //peng --> print every n generation
    //ncp --> number of chromosomes to print
    //if generation is divisable by peng
    if(g%peng==0) {
      //scorefile << "#Generation" << std::setw(14) << "Chromosomes" << std::setw(12) << "Scores\n";
      hipMemcpy(scores, scores_ds[curList], sizeof(*scores)*ncp, hipMemcpyDeviceToHost); //copy over ncp scores
      hipMemcpy(areas, areas_d, sizeof(*areas)*ncp, hipMemcpyDeviceToHost); //copy over ncp areas
      // divide score by the number of datasets to print the average of the datasets since score is sum of each dataset score
      for(int m=0;m<ncp;m++){
        scorefile << std::setw(6) << g << std::setw(14) << m << std::setw(18) << scores[m]/nDataset << std::setw(18) << areas[m] << "\n";
      }
    }
/* END GENETIC ALGORITM */
  } 

  scorefile.close();
/****************************************************************************************************
*    TERMINATION, LAST RESULTS < SCORES AND PARAMETERS FOR EACH INDIVIDUAL
****************************************************************************************************/
 
/***************************************************************************************************/
  /*  copy over the results from GPU to the CPU to save the scores and parameters */
  hipMemcpy(Vs, Vs_d, sizeof(float)*genomeSize*N, hipMemcpyDeviceToHost);
  hipMemcpy(ptrs, ptrs_ds[curList], sizeof(int)*N, hipMemcpyDeviceToHost);
  hipMemcpy(scores, scores_ds[curList], sizeof(float)*N, hipMemcpyDeviceToHost);
  hipMemcpy(tgts, tgts_d, sizeof(float)*nConf, hipMemcpyDeviceToHost);
  hipMemcpy(tset, tset_d, nConf*trainingSize*sizeof(float), hipMemcpyDeviceToHost);

/****************************************************************************************************/
// Here we will move the parameters back into the dihedral space 
   // TODO: Write these out as functions in a seperate files
  // kept the parameters sorted so first Vs in ptrs[i] is best in Vs = Vs_dih[0]
  float *Vs_dih;    
  Vs_dih=(float *)malloc(N*trainingSize*sizeof(float));
  int fg,begv,endv;
  // For a given chromosome (set of Vs)
  for(int i=0;i<N;i++){
    int pt=ptrs[i]; // set the pointer index into the first element of Vs array
    int kN=i*trainingSize;
    // for a given dihedral define in the input file
    int k=0; 
    for (int dih=0;dih<nDih;dih++){
      // get the fitting group it belongs to 
      fg=DihFgindx[dih]; //get the fittting group of that dihedal 
      //std::cout << "fg = " << fg << " for dih = " << dih << std::endl;
      // get the pointers into the Vs for that fg 
      begv=pt+ptrsV[fg];
      endv=begv+nVperFg[fg];
      for (int v=begv;v<endv;v++){
        Vs_dih[kN+k]=Vs[v];
        //printf("begv = %d, endv = %d, v = %d, kN+k = %d, Vs[v] = %f\n", begv,endv,v,kN+k,Vs[v]);
        k++;
      }
    }
  }
/****************************************************************************************************/
   // Here I am writing out the initial dE and the final dE, see load.cpp for description 
  /* file that stores initial dE */
  std::ofstream fitfile;
  fitfile.open (fitFile.c_str(), ios::out); 
  int i0, t;
  int b=0;
  int d=0; // index into dataset
  int c=0; // conformation index
  float DS_score[nDataset]; //hold the dataset scores
  float x[nConf];  // for the error of each conformation
  float *S=scores+0;
  // set score to 0
  *S=0.0f;
  // accumulate little s for each set
  float s;
  int tg,beg,end;
  int tindx;
  int pt = ptrs[0]; //only want the best, is it sorted as yet??? set the pointer index into the first element of Vs array
  fitfile << "DATASET "<< d << ":" << "\n";
  while(c<nConf){
     //s is the sum of REE
     s=0.0f;
     /* loop only over in conformations within a dataset */
     while(c<breaks[b+1]){
        float parm=0;
        /* start with delta E (tgts) for a given conformation (c) within a break; see load.cpp
          conf (c) goes through until it reach a break. the loop will set delta E */
        // get first index in genome
        i0=pt;
        //printf("i0: %d ", i0);
        // get dE for that conformation
        x[c]=tgts[c];
        // Get the number of dihedral in the dataset
        // loop throught the dihedrals of a given conformation
        //printf("ptrsD ??: ptrsD[d] = %d, ptrsD[d+1] = %d, d = %d\n", ptrsD[d],ptrsD[d+1],d);
        tindx=0; //index into the ptrsT array 0 to number of dihedral columns in a given dataset
        float holdc = x[c];
        fitfile << "dE for Conf "<< c << ":  " << holdc;
        for (int dih=ptrsD[d];dih<ptrsD[d+1];dih++,tindx++){
          //Get the fitting group for that dihedral
          fg=allFginDs[dih];
          //printf("Fitting group = %d for dih index %d\n", allFginDs[dih], dih);
          //get the index into Vs and tset
          beg=i0+ptrsV[fg];
          end=beg+nVperFg[fg];
          tg=ptrsT[(c*trainingSize)+tindx]; //index into prtsT
          t=(c*trainingSize)+tg;
          //printf("beg = %d, end = %d, tg = %d, tindx = %d t = %d \n", beg,end,tg,tindx,t);
          //loop through the number of cosines
          for (int i=beg;i<end;i++,t++) {
             /* subtract contributions from each parameter for conformation c for each conformation
             e.g deltaE - cos (dihedral * periodicity) * parameter generated from chromosomes
             Therefore, it is delta E - sum of cosines for each dihedral */
            x[c]-=Vs[i] * tset[t]; // Vs* tset is cos(n * dih)
            parm += Vs[i] * (1+tset[t]);
//#if DEBUG>2
            //printf("scoreIt: i = %d, c = %d, dih = %d, beg = %d, end = %d, t = %d, x[c] = %f,  Vs[i] = %f, tset[t] = %f \n",i,c,dih,beg,end,t,x[c],Vs[i],tset[t]);
//#endif
          }

        }
        fitfile << "; Parameters Energy for Conf "<< c << ": " << parm << "\n";
        fitfile << "; MM0 Energy for Conf "<< c << ": " << EMM0[c] << "\n";
        fitfile << "; MM Energy for Conf "<< c << ": " << (EMM0[c] + parm) << "\n";

        /* add differences in this error from all other errors */
        //printf("outside loopscore for x[c] = %f\n", x[c]);
        for(int c2=breaks[b];c2<c;c2++){
#if DEBUG>2
          printf("In loop score for x[c] = %f\n", x[c]);
          printf("%d - %d\n",c,c2); //print the pairs index
#endif
            // calculate the absolute error for each pairs
          float err=x[c]-x[c2];
            // sum the absolute of the errors (err) - -err = + err ; +err = +err
            //s+=(err<0.0f?-err:err); //ternary operator, condition is err < 0.0; if true err is negative, if false error is positive
          s+=abs(err);
          fitfile << "REE for Conf " << c << " and " << c2 << ": " << abs(err) << "\n";
        }
        //printf("score for c %d = %f\n", c,s);
        /* next conformation */
        ++c;
      }
      /* add little error to big error S, weighted by number of pairs, wt  is 2 / nconf*(nconf-1) */
      *S+=s*wts[b];
      DS_score[d] = s*wts[b];
      /* go to next breakpoint (data set) */
      ++b;
      ++d;
  }
  
  fitfile << "Scores per Datasets:" << "\n";
  for(int d=0;d<nDataset;d++){
    fitfile << std::setw(6) << d << std::setw(18) << DS_score[d] << "\n\n";
  } 
  fitfile.close();
/****************************************************************************************************/

  /* saving all of the scores, with dihedral parameters to the logfile */
  logfile << "\n";
  logfile << "Printing all of the final dihedral parameters, check your -f file for the best one \n\n";
  logfile << "The first one is the best score, best parameters\n\n";
  /* loop through the population */
  for(int i=0;i<pSize;i++){
    // these are the final scores for each individual in the population, print in the output file  
    // divide score by the number of datasets to print the average of the datasets since score is sum of each dataset score
    logfile << std::fixed << "chromosome: " << ptrs[i]/genomeSize << std::endl;
    logfile << std::fixed << "Average Score: " << scores[i]/nDataset << std::endl;
    for(std::map<std::string,DihCorrection>::iterator it=correctionMap.begin(); it!=correctionMap.end(); ++it){
    // second.setGenome(Vs+ptrs[i]) is the dihedral parameters for each individual in the population 
      //print in the output file                                                                    
      //logfile << it->second.setGenome(Vs+ptrs[i]);
      logfile << it->second.setGenome(Vs_dih+(i*trainingSize));
    }
  }
/****************************************************************************************************/
  /* Save a frcmod file to use in Amber */

  if(!frcmodFile.empty()){
    std::ofstream frcmodfile;
    frcmodfile.open (frcmodFile.c_str(), ios::out);
    frcmodfile << "frcmod from GenA.cu \n";
    frcmodfile << "DIHE\n";
    int holdFG[nFg] = {-1};

    // loop through all dihedral DihCorrection map (this is the dihedrals names/atomtypes in the input file)
    for(std::map<std::string,DihCorrection>::iterator it=correctionMap.begin(); it!=correctionMap.end(); ++it){
      // loop through fitting groups to check if this fitting group is already printed 
      for(int f=0;f<nFg;f++){
        // if it is not already printed 
        if (holdFG[f] != f) { 
          if (it->second.fitgrpindx == f) {
             //frcmodfile << it->first << "\n"; // dihedral name
             frcmodfile << it->second.setGenome(Vs_dih+0); //the best parameters  
             holdFG[f]=f;
          } 
        }
      }
    }
    frcmodfile.close();
  }
/****************************************************************************************************/
  /* Save the amplitudes to a restart file  */
  if(!saveFile.empty()){
    std::ofstream savefile;
    savefile.open (saveFile.c_str(), ios::out);
    // Write restart in parameter space 
    for(int i=0;i<N;i++){
      for(int j=0;j<genomeSize;j++){
        //savefile << std::setw(9) << ptrs[i]+j << " ";
        savefile << std::setw(9) << Vs[ptrs[i]+j] << " ";
      }
      savefile <<"\n";
    }

    // write to file in dihedral space 
    savefile <<"\n\n\n\n\n";
    for(int i=0;i<N;i++){
      int kN=i*trainingSize;
      for(int j=0;j<trainingSize;j++){
        savefile << std::setw(9) << Vs_dih[kN+j] << " ";
      }
      savefile << "\n";
    } 
  savefile.close();
  }

/****************************************************************************************************/
  //END timing and report time in log file
  auto t2=std::chrono::high_resolution_clock::now();
  logfile <<"\n\n";
  logfile << "RAGTAG took " 
          << std::chrono::duration_cast<std::chrono::milliseconds>(t2-t1).count() 
          << " milli seconds to obtain your parameters" << "\n";

  logfile.close(); //close log file

/*****************| Free up Memory |*******************************************************/
  free(ptrs);
  hiprandDestroyGenerator(gen);
  //hipFree(xx_d);
  hipFree(Vs_d);
  hipFree(ptrs_d);
  hipFree(breaks_d);
  hipFree(tgts_d);
  free(Vs);
  free(scores);
  //hipFree(rands_d);
  free(rands);
  return 0;
}
